#include "hip/hip_runtime.h"
#include "kernel.h"
#include "hip/hip_runtime.h"
#include ""
#include<thrust/complex.h>
 #define TX 32
 #define TY 32
#define max_iteration 500

 __device__
 unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

 __global__
 void distanceKernel(uchar4 * d_out, int w, int h, int2 pos) {
	const int c = blockIdx.x * blockDim.x + threadIdx.x;
	const int r = blockIdx.y * blockDim.y + threadIdx.y;
	if ((c >= w) || (r >= h)) return; // Check if within image bounds
	const int i = c + r * w; // 1D indexing
	const int dist = sqrtf((c - pos.x) * (c - pos.x) +
	 (r - pos.y) * (r - pos.y));
	 const unsigned char intensity = clip(255 - dist);
	 d_out[i].x = intensity;
	 d_out[i].y = intensity;
	 d_out[i].z = 0;
	 d_out[i].w = 255;
	
}
 __global__ void juliaset(uchar4* d_out,int w , int h,thrust::complex<double> c, float zoom ,  double2 offset) {
	 const int o = blockIdx.x * blockDim.x + threadIdx.x;
	 const int r = blockIdx.y * blockDim.y + threadIdx.y;
	 int2 cen = { w / 2,h / 2 };
	 
	 if ((o >= w) || (r >= h)) return; // Check if within image bounds
	 const int i = o + r * w; // 1D indexing
	 thrust::complex<double > pos = { (zoom * 3 * (double(o - cen.x) / w)) + offset.x,(zoom * 3 * (double(r - cen.y) / h)) + offset.y };
	 thrust::complex<double> z = { pos.real(),pos.imag()};
	 
	 int j = 0;
	 for ( j; j < max_iteration; j++) {
		 if (thrust::abs(z ) > 4) break;
		 z =  z*z + c;
		 
	 }
	 double smooth = (double(j) - log2(fmax(1.0, log2(thrust::abs(z ))))) / max_iteration;
	 unsigned char intensity = char(255*smooth);
	 
	 d_out[i].x = intensity;
	 d_out[i].y = intensity;
	 d_out[i].z =intensity;
	 d_out[i].w = 255;
 }
 void kernelLauncher(uchar4 * d_out, int w, int h, double2 pos, float zoom,double2 offset) {
	 const dim3 blockSize(TX, TY);
	 const dim3 gridSize = dim3((w + TX - 1) / TX, (h + TY - 1) / TY);
	 thrust::complex<double> c = { pos.x,pos.y };
	 juliaset << <gridSize, blockSize >> > (d_out, w, h, c, zoom, offset);
	 hipDeviceSynchronize();
	
}
